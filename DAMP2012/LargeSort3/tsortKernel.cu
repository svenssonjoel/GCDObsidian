
#include <hip/hip_runtime.h>
 
__global__ void tsortSmall(int *input0,int *result0){
  unsigned int tid = threadIdx.x;
  unsigned int bid = blockIdx.x;
  extern __shared__  unsigned char sbase[];
  (( int *)sbase)[(tid<<1)] = min(input0[((bid*512)+(tid<<1))],input0[((bid*512)+((tid<<1)^1))]);
  (( int *)sbase)[((tid<<1)^1)] = max(input0[((bid*512)+(tid<<1))],input0[((bid*512)+((tid<<1)^1))]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967294))] = min((( int *)sbase)[(tid+(tid&4294967294))],(( int *)sbase)[((tid+(tid&4294967294))^3)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967294))^3)] = max((( int *)sbase)[(tid+(tid&4294967294))],(( int *)sbase)[((tid+(tid&4294967294))^3)]);
  __syncthreads();
  (( int *)sbase)[(tid<<1)] = min((( int *)(sbase+2048))[(tid<<1)],(( int *)(sbase+2048))[((tid<<1)^1)]);
  (( int *)sbase)[((tid<<1)^1)] = max((( int *)(sbase+2048))[(tid<<1)],(( int *)(sbase+2048))[((tid<<1)^1)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967292))] = min((( int *)sbase)[(tid+(tid&4294967292))],(( int *)sbase)[((tid+(tid&4294967292))^7)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967292))^7)] = max((( int *)sbase)[(tid+(tid&4294967292))],(( int *)sbase)[((tid+(tid&4294967292))^7)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967294))] = min((( int *)(sbase+2048))[(tid+(tid&4294967294))],(( int *)(sbase+2048))[((tid+(tid&4294967294))^2)]);
  (( int *)sbase)[((tid+(tid&4294967294))^2)] = max((( int *)(sbase+2048))[(tid+(tid&4294967294))],(( int *)(sbase+2048))[((tid+(tid&4294967294))^2)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid<<1)] = min((( int *)sbase)[(tid<<1)],(( int *)sbase)[((tid<<1)^1)]);
  (( int *)(sbase + 2048))[((tid<<1)^1)] = max((( int *)sbase)[(tid<<1)],(( int *)sbase)[((tid<<1)^1)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967288))] = min((( int *)(sbase+2048))[(tid+(tid&4294967288))],(( int *)(sbase+2048))[((tid+(tid&4294967288))^15)]);
  (( int *)sbase)[((tid+(tid&4294967288))^15)] = max((( int *)(sbase+2048))[(tid+(tid&4294967288))],(( int *)(sbase+2048))[((tid+(tid&4294967288))^15)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967292))] = min((( int *)sbase)[(tid+(tid&4294967292))],(( int *)sbase)[((tid+(tid&4294967292))^4)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967292))^4)] = max((( int *)sbase)[(tid+(tid&4294967292))],(( int *)sbase)[((tid+(tid&4294967292))^4)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967294))] = min((( int *)(sbase+2048))[(tid+(tid&4294967294))],(( int *)(sbase+2048))[((tid+(tid&4294967294))^2)]);
  (( int *)sbase)[((tid+(tid&4294967294))^2)] = max((( int *)(sbase+2048))[(tid+(tid&4294967294))],(( int *)(sbase+2048))[((tid+(tid&4294967294))^2)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid<<1)] = min((( int *)sbase)[(tid<<1)],(( int *)sbase)[((tid<<1)^1)]);
  (( int *)(sbase + 2048))[((tid<<1)^1)] = max((( int *)sbase)[(tid<<1)],(( int *)sbase)[((tid<<1)^1)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967280))] = min((( int *)(sbase+2048))[(tid+(tid&4294967280))],(( int *)(sbase+2048))[((tid+(tid&4294967280))^31)]);
  (( int *)sbase)[((tid+(tid&4294967280))^31)] = max((( int *)(sbase+2048))[(tid+(tid&4294967280))],(( int *)(sbase+2048))[((tid+(tid&4294967280))^31)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967288))] = min((( int *)sbase)[(tid+(tid&4294967288))],(( int *)sbase)[((tid+(tid&4294967288))^8)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967288))^8)] = max((( int *)sbase)[(tid+(tid&4294967288))],(( int *)sbase)[((tid+(tid&4294967288))^8)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967292))] = min((( int *)(sbase+2048))[(tid+(tid&4294967292))],(( int *)(sbase+2048))[((tid+(tid&4294967292))^4)]);
  (( int *)sbase)[((tid+(tid&4294967292))^4)] = max((( int *)(sbase+2048))[(tid+(tid&4294967292))],(( int *)(sbase+2048))[((tid+(tid&4294967292))^4)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967294))] = min((( int *)sbase)[(tid+(tid&4294967294))],(( int *)sbase)[((tid+(tid&4294967294))^2)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967294))^2)] = max((( int *)sbase)[(tid+(tid&4294967294))],(( int *)sbase)[((tid+(tid&4294967294))^2)]);
  __syncthreads();
  (( int *)sbase)[(tid<<1)] = min((( int *)(sbase+2048))[(tid<<1)],(( int *)(sbase+2048))[((tid<<1)^1)]);
  (( int *)sbase)[((tid<<1)^1)] = max((( int *)(sbase+2048))[(tid<<1)],(( int *)(sbase+2048))[((tid<<1)^1)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967264))] = min((( int *)sbase)[(tid+(tid&4294967264))],(( int *)sbase)[((tid+(tid&4294967264))^63)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967264))^63)] = max((( int *)sbase)[(tid+(tid&4294967264))],(( int *)sbase)[((tid+(tid&4294967264))^63)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967280))] = min((( int *)(sbase+2048))[(tid+(tid&4294967280))],(( int *)(sbase+2048))[((tid+(tid&4294967280))^16)]);
  (( int *)sbase)[((tid+(tid&4294967280))^16)] = max((( int *)(sbase+2048))[(tid+(tid&4294967280))],(( int *)(sbase+2048))[((tid+(tid&4294967280))^16)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967288))] = min((( int *)sbase)[(tid+(tid&4294967288))],(( int *)sbase)[((tid+(tid&4294967288))^8)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967288))^8)] = max((( int *)sbase)[(tid+(tid&4294967288))],(( int *)sbase)[((tid+(tid&4294967288))^8)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967292))] = min((( int *)(sbase+2048))[(tid+(tid&4294967292))],(( int *)(sbase+2048))[((tid+(tid&4294967292))^4)]);
  (( int *)sbase)[((tid+(tid&4294967292))^4)] = max((( int *)(sbase+2048))[(tid+(tid&4294967292))],(( int *)(sbase+2048))[((tid+(tid&4294967292))^4)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967294))] = min((( int *)sbase)[(tid+(tid&4294967294))],(( int *)sbase)[((tid+(tid&4294967294))^2)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967294))^2)] = max((( int *)sbase)[(tid+(tid&4294967294))],(( int *)sbase)[((tid+(tid&4294967294))^2)]);
  __syncthreads();
  (( int *)sbase)[(tid<<1)] = min((( int *)(sbase+2048))[(tid<<1)],(( int *)(sbase+2048))[((tid<<1)^1)]);
  (( int *)sbase)[((tid<<1)^1)] = max((( int *)(sbase+2048))[(tid<<1)],(( int *)(sbase+2048))[((tid<<1)^1)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967232))] = min((( int *)sbase)[(tid+(tid&4294967232))],(( int *)sbase)[((tid+(tid&4294967232))^127)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967232))^127)] = max((( int *)sbase)[(tid+(tid&4294967232))],(( int *)sbase)[((tid+(tid&4294967232))^127)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967264))] = min((( int *)(sbase+2048))[(tid+(tid&4294967264))],(( int *)(sbase+2048))[((tid+(tid&4294967264))^32)]);
  (( int *)sbase)[((tid+(tid&4294967264))^32)] = max((( int *)(sbase+2048))[(tid+(tid&4294967264))],(( int *)(sbase+2048))[((tid+(tid&4294967264))^32)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967280))] = min((( int *)sbase)[(tid+(tid&4294967280))],(( int *)sbase)[((tid+(tid&4294967280))^16)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967280))^16)] = max((( int *)sbase)[(tid+(tid&4294967280))],(( int *)sbase)[((tid+(tid&4294967280))^16)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967288))] = min((( int *)(sbase+2048))[(tid+(tid&4294967288))],(( int *)(sbase+2048))[((tid+(tid&4294967288))^8)]);
  (( int *)sbase)[((tid+(tid&4294967288))^8)] = max((( int *)(sbase+2048))[(tid+(tid&4294967288))],(( int *)(sbase+2048))[((tid+(tid&4294967288))^8)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967292))] = min((( int *)sbase)[(tid+(tid&4294967292))],(( int *)sbase)[((tid+(tid&4294967292))^4)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967292))^4)] = max((( int *)sbase)[(tid+(tid&4294967292))],(( int *)sbase)[((tid+(tid&4294967292))^4)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967294))] = min((( int *)(sbase+2048))[(tid+(tid&4294967294))],(( int *)(sbase+2048))[((tid+(tid&4294967294))^2)]);
  (( int *)sbase)[((tid+(tid&4294967294))^2)] = max((( int *)(sbase+2048))[(tid+(tid&4294967294))],(( int *)(sbase+2048))[((tid+(tid&4294967294))^2)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid<<1)] = min((( int *)sbase)[(tid<<1)],(( int *)sbase)[((tid<<1)^1)]);
  (( int *)(sbase + 2048))[((tid<<1)^1)] = max((( int *)sbase)[(tid<<1)],(( int *)sbase)[((tid<<1)^1)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967168))] = min((( int *)(sbase+2048))[(tid+(tid&4294967168))],(( int *)(sbase+2048))[((tid+(tid&4294967168))^255)]);
  (( int *)sbase)[((tid+(tid&4294967168))^255)] = max((( int *)(sbase+2048))[(tid+(tid&4294967168))],(( int *)(sbase+2048))[((tid+(tid&4294967168))^255)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967232))] = min((( int *)sbase)[(tid+(tid&4294967232))],(( int *)sbase)[((tid+(tid&4294967232))^64)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967232))^64)] = max((( int *)sbase)[(tid+(tid&4294967232))],(( int *)sbase)[((tid+(tid&4294967232))^64)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967264))] = min((( int *)(sbase+2048))[(tid+(tid&4294967264))],(( int *)(sbase+2048))[((tid+(tid&4294967264))^32)]);
  (( int *)sbase)[((tid+(tid&4294967264))^32)] = max((( int *)(sbase+2048))[(tid+(tid&4294967264))],(( int *)(sbase+2048))[((tid+(tid&4294967264))^32)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967280))] = min((( int *)sbase)[(tid+(tid&4294967280))],(( int *)sbase)[((tid+(tid&4294967280))^16)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967280))^16)] = max((( int *)sbase)[(tid+(tid&4294967280))],(( int *)sbase)[((tid+(tid&4294967280))^16)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967288))] = min((( int *)(sbase+2048))[(tid+(tid&4294967288))],(( int *)(sbase+2048))[((tid+(tid&4294967288))^8)]);
  (( int *)sbase)[((tid+(tid&4294967288))^8)] = max((( int *)(sbase+2048))[(tid+(tid&4294967288))],(( int *)(sbase+2048))[((tid+(tid&4294967288))^8)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967292))] = min((( int *)sbase)[(tid+(tid&4294967292))],(( int *)sbase)[((tid+(tid&4294967292))^4)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967292))^4)] = max((( int *)sbase)[(tid+(tid&4294967292))],(( int *)sbase)[((tid+(tid&4294967292))^4)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967294))] = min((( int *)(sbase+2048))[(tid+(tid&4294967294))],(( int *)(sbase+2048))[((tid+(tid&4294967294))^2)]);
  (( int *)sbase)[((tid+(tid&4294967294))^2)] = max((( int *)(sbase+2048))[(tid+(tid&4294967294))],(( int *)(sbase+2048))[((tid+(tid&4294967294))^2)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid<<1)] = min((( int *)sbase)[(tid<<1)],(( int *)sbase)[((tid<<1)^1)]);
  (( int *)(sbase + 2048))[((tid<<1)^1)] = max((( int *)sbase)[(tid<<1)],(( int *)sbase)[((tid<<1)^1)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967040))] = min((( int *)(sbase+2048))[(tid+(tid&4294967040))],(( int *)(sbase+2048))[((tid+(tid&4294967040))^511)]);
  (( int *)sbase)[((tid+(tid&4294967040))^511)] = max((( int *)(sbase+2048))[(tid+(tid&4294967040))],(( int *)(sbase+2048))[((tid+(tid&4294967040))^511)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967168))] = min((( int *)sbase)[(tid+(tid&4294967168))],(( int *)sbase)[((tid+(tid&4294967168))^128)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967168))^128)] = max((( int *)sbase)[(tid+(tid&4294967168))],(( int *)sbase)[((tid+(tid&4294967168))^128)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967232))] = min((( int *)(sbase+2048))[(tid+(tid&4294967232))],(( int *)(sbase+2048))[((tid+(tid&4294967232))^64)]);
  (( int *)sbase)[((tid+(tid&4294967232))^64)] = max((( int *)(sbase+2048))[(tid+(tid&4294967232))],(( int *)(sbase+2048))[((tid+(tid&4294967232))^64)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967264))] = min((( int *)sbase)[(tid+(tid&4294967264))],(( int *)sbase)[((tid+(tid&4294967264))^32)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967264))^32)] = max((( int *)sbase)[(tid+(tid&4294967264))],(( int *)sbase)[((tid+(tid&4294967264))^32)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967280))] = min((( int *)(sbase+2048))[(tid+(tid&4294967280))],(( int *)(sbase+2048))[((tid+(tid&4294967280))^16)]);
  (( int *)sbase)[((tid+(tid&4294967280))^16)] = max((( int *)(sbase+2048))[(tid+(tid&4294967280))],(( int *)(sbase+2048))[((tid+(tid&4294967280))^16)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967288))] = min((( int *)sbase)[(tid+(tid&4294967288))],(( int *)sbase)[((tid+(tid&4294967288))^8)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967288))^8)] = max((( int *)sbase)[(tid+(tid&4294967288))],(( int *)sbase)[((tid+(tid&4294967288))^8)]);
  __syncthreads();
  (( int *)sbase)[(tid+(tid&4294967292))] = min((( int *)(sbase+2048))[(tid+(tid&4294967292))],(( int *)(sbase+2048))[((tid+(tid&4294967292))^4)]);
  (( int *)sbase)[((tid+(tid&4294967292))^4)] = max((( int *)(sbase+2048))[(tid+(tid&4294967292))],(( int *)(sbase+2048))[((tid+(tid&4294967292))^4)]);
  __syncthreads();
  (( int *)(sbase + 2048))[(tid+(tid&4294967294))] = min((( int *)sbase)[(tid+(tid&4294967294))],(( int *)sbase)[((tid+(tid&4294967294))^2)]);
  (( int *)(sbase + 2048))[((tid+(tid&4294967294))^2)] = max((( int *)sbase)[(tid+(tid&4294967294))],(( int *)sbase)[((tid+(tid&4294967294))^2)]);
  __syncthreads();
  (( int *)sbase)[(tid<<1)] = min((( int *)(sbase+2048))[(tid<<1)],(( int *)(sbase+2048))[((tid<<1)^1)]);
  (( int *)sbase)[((tid<<1)^1)] = max((( int *)(sbase+2048))[(tid<<1)],(( int *)(sbase+2048))[((tid<<1)^1)]);
  __syncthreads();
  result0[((bid*512)+tid)] = (( int *)sbase)[tid];
  result0[((bid*512)+(tid+256))] = (( int *)sbase)[(tid+256)];
  
}